#include "hip/hip_runtime.h"
#include "Vec3.h"
#include "Camera.h"
#include "chrono"

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "third-party/stb_image_write.h"
#include "third-party/BS_thread_pool.h"

#include "third-party/BS_thread_pool.h"

#define TINYOBJLOADER_IMPLEMENTATION

#include "third-party/tiny_obj_loader.h"

#include "Scene.h"

#include "hiprand.h"
#include "cmath"

int main(int argc, char *argv[]) {
    int image_width = 256;
    int image_height = 256;
    int samples_per_pixel = 1;
    int num_threads = 1;
    if (argc != 1) {
        if (argc > 1) {
            samples_per_pixel = std::atoi(argv[1]);
        }
        if (argc > 2) {
            num_threads = std::atoi(argv[2]);
        }
        if (argc > 3) {
            image_width = std::atoi(argv[3]) * 16 / 9;
            image_height = std::atoi(argv[3]);
        }
    }

    unsigned char pixels[image_width * image_height * 4];
    auto camera = HoloCamera(image_width, image_height, samples_per_pixel, 10);

    hiprandCreateGeneratorHost(&Random::l_rand, hiprandRngType_t::HIPRAND_RNG_PSEUDO_DEFAULT);

    HittableList *world;
    world = CPUScene::hologram(camera);


    BS::thread_pool pool{static_cast<unsigned int>(num_threads)};

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << camera.samples_per_pixel
              << " samples per pixel " << "with " << pool.get_thread_count() << " threads.\n";

    auto start = time(nullptr);

    auto *pixels_complex = new std::complex<double>[image_width * image_height];
    camera.render_CGH(pixels_complex, &world);

    for (int i = 0; i < image_width * image_height; i++) {
        pixels[i * 4 + 0] = std::arg(pixels_complex[i]) / (2 * M_PI) * 255;
        pixels[i * 4 + 1] = std::arg(pixels_complex[i]) / (2 * M_PI) * 255;
        pixels[i * 4 + 2] = std::arg(pixels_complex[i]) / (2 * M_PI) * 255;
        pixels[i * 4 + 3] = 255;
    }

    /*
    pool.detach_loop(0, image_height, [camera, &pixels, &world](int j) {
        camera.render_pixel_line(&pixels[j * camera.image_width * 4], &world, (int) j);
    }, 50);
    pool.wait();
    */
    auto end = time(nullptr);
    auto duration = end - start;

    stbi_write_png(string_format("ph_%dx%d_%d_%d_%dcpu_%.1ld.png", image_height, image_width,
                                 camera.samples_per_pixel, camera.max_depth, pool.get_thread_count(), duration).c_str(),
                   image_width, image_height, 4, pixels, 0);

    std::cerr << "Rendered in " << duration << "s" << std::endl;

    auto min = std::abs(
            *std::min_element(pixels_complex, pixels_complex + image_width * image_height, [](auto a, auto b) {
                return std::abs(a) < std::abs(b);
            }));

    auto max = std::abs(
            *std::max_element(pixels_complex, pixels_complex + image_width * image_height, [](auto a, auto b) {
                return std::abs(a) < std::abs(b);
            }));

    for (int i = 0; i < image_width * image_height; i++) {
        pixels[i * 4 + 0] = static_cast<unsigned char>((std::abs(pixels_complex[i]) - min) / (max - min) * 255);
        pixels[i * 4 + 1] = static_cast<unsigned char>((std::abs(pixels_complex[i]) - min) / (max - min) * 255);
        pixels[i * 4 + 2] = static_cast<unsigned char>((std::abs(pixels_complex[i]) - min) / (max - min) * 255);
        pixels[i * 4 + 3] = 255;
    }

    stbi_write_png(string_format("amp_%dx%d_%d_%d_%dcpu_%.1ld.png", image_height, image_width,
                                 camera.samples_per_pixel, camera.max_depth, pool.get_thread_count(), duration).c_str(),
                   image_width, image_height, 4, pixels, 0);


}