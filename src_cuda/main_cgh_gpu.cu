#include "hip/hip_runtime.h"
#define SCREEN_HEIGHT_IN_PX 50
#define ENABLE_RANDOM_SCREEN_RAYS

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "third-party/stb_image_write.h"

#define TINYOBJLOADER_IMPLEMENTATION

#include "third-party/tiny_obj_loader.h"

#include "third-party/BS_thread_pool.h"

#include "utils.h"
#include "hip/hip_runtime.h"
#include <chrono>
#include <cfloat>
#include <hiprand/hiprand_kernel.h>
#include <thrust/complex.h>

#include "Vec3.h"
#include "Camera.h"
#include "hittable/Sphere.h"
#include "Scene.h"
#include "third-party/argparse.hpp"

__global__ void
render(thrust::complex<double> *fb, int max_x, int max_y, HoloCamera *d_camera, HittableList **d_world,
       Point3 *point_cloud,
       int point_cloud_size, hiprandState *global_state) {

    uint i = threadIdx.x + blockIdx.x * blockDim.x;
    uint j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    uint pixel_index = j * max_x + i;
    hiprandState local_state = global_state[pixel_index];

    const auto *world = *d_world;
    auto slm_pixel_center =
            d_camera->slm_pixel_00_location + (i * d_camera->slm_pixel_delta_x) + (j * d_camera->slm_pixel_delta_y);

    for (int pi = 0; pi < point_cloud_size; pi++) {
        auto ray = Ray(slm_pixel_center, point_cloud[pi] - slm_pixel_center);
        const thrust::complex<double> cgh = d_camera->ray_wave_cgh(ray, point_cloud[pi], d_camera->max_depth, *world,
                                                                   &local_state);
        fb[pixel_index] += cgh;
    }
    fb[pixel_index] /= (d_camera->slm_width_in_px * d_camera->slm_height_in_px * 1.0);
}

__global__ void render_init(int max_x, int max_y, hiprandState *global_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;

    hiprand_init(pixel_index, 0, 0, &global_state[pixel_index]);
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(0, 0, 0, rand_state);
    }
}

struct MyArgs : public argparse::Args {
    int &image_height = kwarg("ih,image-height", "Image height in pixels").set_default(1080);
    int &screen_height = kwarg("sh,screen-height", "Screen height in pixels (for the point cloud)").set_default(SCREEN_HEIGHT_IN_PX);
    int &samples_per_pixel = kwarg("spp,samples", "Number of samples per pixel (unimplemented)").set_default(1);
    int &max_depth = kwarg("d,depth", "Maximum ray depth").set_default(10);
    int &tx = kwarg("tx", "Number of blocks in the x dimension").set_default(32);
    int &ty = kwarg("ty", "Number of blocks in the y dimension").set_default(16);
    bool &verbose = flag("v,verbose", "A flag to toggle verbose");
};

int main(int argc, char *argv[]) {
    auto args = argparse::parse<MyArgs>(argc, argv);

    const int image_width = args.image_height * 16 / 9;
    const int image_height = args.image_height;
    const int screen_height = args.screen_height;

    const int samples_per_pixel = args.samples_per_pixel;
    const int max_depth = args.max_depth;

    const int tx = args.tx;
    const int ty = args.ty;

    if (args.verbose) {
        args.print();
    }

    unsigned char pixels[image_width * image_height];

    auto start = time(nullptr);

    auto num_pixels = image_width * image_height;
    auto frame_buffer_size = num_pixels * sizeof(thrust::complex<double>);

    dim3 block(tx, ty);
    dim3 grid((image_width + block.x - 1) / block.x, (image_height + block.y - 1) / block.y);


    thrust::complex<double> *fb;
    hiprandState *d_global_state;
    hiprandState *d_global_state2;
    CU(hipMalloc((void **) &d_global_state2, 1 * sizeof(hiprandState)));
    rand_init<<<1, 1>>>(d_global_state2);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());

    Hittable **d_list;
    HittableList **d_world;
    HoloCamera *d_camera;

    d_camera = new(true) HoloCamera(image_width, image_height, samples_per_pixel, max_depth, screen_height);

    Scene::hologram(&d_list, &d_world, *d_camera, d_global_state2);
    const HittableList *h_world = Scene::hologram_cpu(*d_camera);

    auto point_cloud = d_camera->generate_point_cloud(*h_world);

    Vec3 *d_point_cloud;

    CU(hipMalloc((void **) &d_point_cloud, point_cloud.size() * sizeof(Vec3)));
    CU(hipMemcpy(d_point_cloud, point_cloud.data(), point_cloud.size() * sizeof(Vec3), hipMemcpyHostToDevice));

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << d_camera->samples_per_pixel
              << " samples per pixel " << "in " << tx << "x" << ty << " blocks.\n";

    CU(hipMallocManaged((void **) &fb, frame_buffer_size));
    CU(hipMalloc((void **) &d_global_state, num_pixels * sizeof(hiprandState)));

    render_init<<<grid, block>>>(image_width, image_height, d_global_state);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());

    render<<<grid, block>>>(fb, image_width, image_height,
                            d_camera,
                            d_world,
                            d_point_cloud,
                            point_cloud.size(),
                            d_global_state);
    CU(hipGetLastError());
    CU(hipDeviceSynchronize());

// std::arg -> If no errors occur, this is the phase angle of z in the interval [−π; π].
// [-pi, pi] -> [0, 2pi] -> [0, 1] -> [0, 255]
    for (int i = 0; i < image_width * image_height; i++) {
        pixels[i] = static_cast<unsigned char>((thrust::arg(fb[i]) + M_PI) / (2 * M_PI) * 255);
    }

    auto end = time(nullptr);
    auto duration = end - start;

    std::string filename;

    if (image_width == 1920 && image_height == 1080 && d_camera->samples_per_pixel == 1 && d_camera->max_depth == 10) {
        filename = string_format("ph_%dgpu_%.1lds.png", THREADS, duration);
    } else {
        filename = string_format("ph_%dx%d_%d_%d_%dgpu_%.1ld.png", image_height, image_width,
                                 d_camera->samples_per_pixel, d_camera->max_depth, THREADS,
                                 duration);
    }
    stbi_write_png(filename.c_str(), image_width, image_height, 1, pixels, 0);
    std::cerr << "Rendered in " << duration << "s" << std::endl;
    printf("Image saved as: %s\n", filename.c_str());

    auto command = string_format("python3 ../propagation/main.py %s", filename.c_str());
    std::system(command.c_str());
}
